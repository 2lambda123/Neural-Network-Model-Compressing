#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/inq_conv_layer.hpp"
#include <cmath>

namespace caffe {

template <typename Dtype>
__global__ void TPCalc(const int n, Dtype *param, Dtype *mask,
                       const Dtype threshold_, const int max_quantum_exp_,
                       const int min_quantum_exp_) {
  CUDA_KERNEL_LOOP(i, n) {
    if (mask[i] == 1) {
      if (param[i] >= threshold_) {
        // exp_ won't be larger than max_quantum_exp_, already checked in the
        int exp_ = floor(log(4.0 * param[i] / 3.0) / log(2.0));
        // CHECK_LE(exp_, max_quantum_exp_) ;
        if (exp_ >= min_quantum_exp_) {
          param[i] = pow(2.0, exp_);
        } else {
          param[i] = 0;
        }
        mask[i] = 0;
      } else if (param[i] <= -threshold_) {
        int exp_ = floor(log(4.0 * (-param[i]) / 3.0) / log(2.0));
        if (exp_ >= min_quantum_exp_) {
          param[i] = -pow(2.0, exp_);
        } else {
          param[i] = 0;
        }
        mask[i] = 0;
      }
    }
  }
}

template <typename Dtype>
__global__ void CCMaskApply(const int n, const Dtype *wb, const Dtype *mask,
                            Dtype *wb_t) {
  CUDA_KERNEL_LOOP(index, n) { wb_t[index] = wb[index] * mask[index]; }
}

template <typename Dtype>
void INQConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype> *> &bottom, const vector<Blob<Dtype> *> &top) {
  /* for two-power network */
  if (this->phase_ == TRAIN) {
    if (this->iter_ == 0) {
      // Make the corresponding weights & bias into two power form.
      if (this->blobs_.size() == 4 && (this->bias_term_)) {
        LOG(INFO) << "Shaping the weights in tp_conv...[gpu]";
        ComputeQuantumRange(this->blobs_[0].get(), this->blobs_[2].get(),
                            this->portions_, weight_quantum_values_,
                            num_weight_quantum_values_, max_weight_quantum_exp_,
                            min_weight_quantum_exp_);
        ShapeIntoTwoPower(this->blobs_[0].get(), this->blobs_[2].get(),
                          this->portions_, max_weight_quantum_exp_,
                          min_weight_quantum_exp_);
        LOG(INFO) << "Shaping the bias in tp_conv...[gpu]";
        ComputeQuantumRange(this->blobs_[1].get(), this->blobs_[3].get(),
                            this->portions_, bias_quantum_values_,
                            num_bias_quantum_values_, max_bias_quantum_exp_,
                            min_bias_quantum_exp_);
        ShapeIntoTwoPower(this->blobs_[1].get(), this->blobs_[3].get(),
                          this->portions_, max_bias_quantum_exp_,
                          min_bias_quantum_exp_);
        LOG(INFO) << "Shaping done in tp_conv...[gpu]";
      } else if (this->blobs_.size() == 2 && (!this->bias_term_)) {
        LOG(INFO) << "ERROR: No bias terms found... but continue...";
        std::cout << "Shaping ONLY the weights..." << std::endl;
        ComputeQuantumRange(this->blobs_[0].get(), this->blobs_[1].get(),
                            this->portions_, weight_quantum_values_,
                            num_weight_quantum_values_, max_weight_quantum_exp_,
                            min_weight_quantum_exp_);
        ShapeIntoTwoPower(this->blobs_[0].get(), this->blobs_[1].get(),
                          this->portions_, max_weight_quantum_exp_,
                          min_weight_quantum_exp_);
      }
    }
  }

  const Dtype *weight = this->blobs_[0]->mutable_gpu_data();
  const Dtype *bias = NULL;
  if (this->bias_term_) {
    bias = this->blobs_[1]->mutable_gpu_data();
  }

  // Forward calculation with (masked) weight and bias
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype *bottom_data = bottom[i]->gpu_data();
    Dtype *top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + bottom[i]->offset(n), weight,
                             top_data + top[i]->offset(n));
      if (this->bias_term_) {
        this->forward_gpu_bias(top_data + top[i]->offset(n), bias);
      }
    }
  }
}

template <typename Dtype>
void INQConvolutionLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype> *> &top, const vector<bool> &propagate_down,
    const vector<Blob<Dtype> *> &bottom) {
  // LOG(INFO) << "Starting Backward in tp_conv... [gpu]" ;
  const Dtype *weight = this->blobs_[0]->mutable_gpu_data();
  const Dtype *weightMask = this->blobs_[2]->gpu_data();
  Dtype *weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype *top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      const Dtype *biasMask = this->blobs_[3]->gpu_data();
      Dtype *bias_diff = this->blobs_[1]->mutable_gpu_diff();

      CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
                           CAFFE_CUDA_NUM_THREADS>>>(
          this->blobs_[3]->count(), bias_diff, biasMask, bias_diff);
      CUDA_POST_KERNEL_CHECK;

      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + top[i]->offset(n));
      }
      // LOG(INFO) << "bias_diff Backwarded in tp_conv... [gpu]";
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype *bottom_data = bottom[i]->gpu_data();
      Dtype *bottom_diff = bottom[i]->mutable_gpu_diff();

      CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[2]->count()),
                           CAFFE_CUDA_NUM_THREADS>>>(
          this->blobs_[2]->count(), weight_diff, weightMask, weight_diff);
      CUDA_POST_KERNEL_CHECK;
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + bottom[i]->offset(n),
                                top_diff + top[i]->offset(n), weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + top[i]->offset(n), weight,
                                  bottom_diff + bottom[i]->offset(n));
        }
      }
    }
  }
  // LOG(INFO) << "Backward finished in tp_conv... [gpu]";
}

template <typename Dtype>
void INQConvolutionLayer<Dtype>::ComputeQuantumRange(
    const Blob<Dtype> *blob, const Blob<Dtype> *blob_mask,
    const vector<float> portions, vector<Dtype> &quantum_values,
    const int &num_quantum_values, int &max_quantum_exp_,
    int &min_quantum_exp_) {

  quantum_values.resize(2 * num_quantum_values + 1);
  const Dtype *values = blob->cpu_data();
  const Dtype *mask = blob_mask->cpu_data();
  Dtype max_value_tobe_quantized = INT_MIN;
  Dtype max_value_quantized = INT_MIN;
  int updated = 0;

  for (unsigned int k = 0; k < blob->count(); ++k) {
    if (mask[k] == 1) {
      if (fabs(values[k]) > max_value_tobe_quantized) {
        max_value_tobe_quantized = fabs(values[k]);
      }
    } else if (mask[k] == 0) {
      if (fabs(values[k]) > max_value_quantized) {
        max_value_quantized = fabs(values[k]);
      }
      ++updated;
    } else {
      LOG(ERROR) << "Mask value is not 0, nor 1, in tp_inner_product_layer";
    }
  }

  if (max_value_quantized != INT_MIN) {
    // normal situation
    CHECK_GT(updated, 0) << "max_value_quantized is not 0.0, but updated is "
                            "0!";
    max_quantum_exp_ = round(log(max_value_quantized) / log(2.0));
    int max_tobe_quantized_exp_ =
        floor(log(4.0 * max_value_tobe_quantized / 3.0) / log(2.0));
    CHECK_GE(max_quantum_exp_, max_tobe_quantized_exp_);
  } else {
    if (updated == 0) {
      // normal situation (nothing quantized yet)
      LOG_IF(INFO, portions_[0] != 0) << "Warning: nothing quantized yet, "
                                         "portion should probably start with "
                                         "0%%!";
      max_quantum_exp_ =
          floor(log(4.0 * max_value_tobe_quantized / 3.0) / log(2.0));
    } else { // DNS model (max_value_quantized ==0 && update != 0)
      max_quantum_exp_ =
          floor(log(4.0 * max_value_tobe_quantized / 3.0) / log(2.0));
    }
  }

  /*
    if (portions[0] == 0) {
      CHECK_EQ(updated, 0) << updated
                           << " updated values while there should be none!";
      max_quantum_exp_ =
          floor(log(4.0 * max_value_tobe_quantized / 3.0) / log(2.0));
    }
    else {
      max_quantum_exp_ = round(log(max_value_quantized) / log(2.0));
      int max_tobe_quantized_exp_ =
          floor(log(4.0 * max_value_tobe_quantized / 3.0) / log(2.0));
      CHECK_LE(max_tobe_quantized_exp_, max_quantum_exp_)
          << "New quantum exp is greater than the one already got!";
    }
  */
  min_quantum_exp_ = max_quantum_exp_ - num_quantum_values + 1;
  std::cout << "Max_power = " << max_quantum_exp_ << std::endl;
  std::cout << "Min_power = " << min_quantum_exp_ << std::endl;
  for (unsigned int k = 0; k < num_quantum_values; ++k) {
    quantum_values[k] = pow(2.0, max_quantum_exp_ - k);
    quantum_values[2 * num_quantum_values - k] = -quantum_values[k];
  }
  quantum_values[num_quantum_values] = 0;
}

template <typename Dtype>
void INQConvolutionLayer<Dtype>::ShapeIntoTwoPower(
    Blob<Dtype> *input_blob, Blob<Dtype> *mask_blob,
    const vector<float> &portions, const int &max_quantum_exp_,
    const int &min_quantum_exp_) {

  const float previous_portion = portions[0];
  const float current_portion = portions[1];
  Dtype *param = input_blob->mutable_gpu_data();
  Dtype *mask = mask_blob->mutable_gpu_data();

  int count = input_blob->count();

  int num_not_yet_quantized = 0;
  vector<Dtype> sorted_param;
  for (int i = 0; i < count; ++i) {
    if (mask[i] == 1) {
      ++num_not_yet_quantized;
      sorted_param.push_back(fabs(param[i]));
    }
  }
  // just an estimation
  int num_init_not_quantized =
      round(Dtype(num_not_yet_quantized) / (1.0 - previous_portion));
  int num_not_tobe_quantized = num_init_not_quantized * (1.0 - current_portion);
  int num_tobe_update = num_not_yet_quantized - num_not_tobe_quantized;

  if (num_tobe_update > 0) {
    sort(sorted_param.begin(), sorted_param.end());
    Dtype threshold_ = sorted_param[num_not_tobe_quantized];
    TPCalc<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, param, mask, threshold_, max_quantum_exp_, min_quantum_exp_);
    CUDA_POST_KERNEL_CHECK;

    LOG(INFO) << "Shaping finished in INQ_conv... [gpu]";
  }
  /*
      for (int i = 0; i < count; ++i) {
        if (mask[i] == 1) {
          if (param[i] >= threshold_) {
            // exp_ won't be larger than max_quantum_exp_, already checked in
     the
            // ComputeQuantumRange()
            int exp_ = floor(log(4.0 * param[i] / 3.0) / log(2.0));
            // CHECK_LE(exp_, max_quantum_exp_) ;
            if (exp_ >= min_quantum_exp_) {
              param[i] = pow(2.0, exp_);
            }
            else {
              param[i] = 0;
            }
            mask[i] = 0;
          }
          else if (param[i] <= -threshold_) {
            int exp_ = floor(log(4.0 * (-param[i]) / 3.0) / log(2.0));
            if (exp_ >= min_quantum_exp_) {
              param[i] = -pow(2.0, exp_);
            }
            else {
              param[i] = 0;
            }
            mask[i] = 0;
          }
        }
      }
  */
}

INSTANTIATE_LAYER_GPU_FUNCS(INQConvolutionLayer);

} // namespace caffe
